#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>

#define BLK 8
#define dimA (10*BLK*10*BLK)
#define dimB (10*BLK*20*BLK)
#define szA (10*BLK*10*BLK*sizeof(double))
#define szB (10*BLK*20*BLK*sizeof(double))
#define A ((const double (*)[10*BLK])a)
#define B ((const double (*)[20*BLK])b)
#define C ((double (*)[20*BLK])c)
#define bx blockIdx.x
#define by blockIdx.y
#define tx threadIdx.x
#define ty threadIdx.y

void init(int n,double *M){
    int i;
    for(i=0;i<n;i++){
        M[i]=(double)rand()/RAND_MAX;
    }
}
void check(int n,double *x,double *y){
    int i;
    double maxerr=0;
    for(i=0;i<n;i++){
        if(fabsf(x[i]-y[i])/y[i]>maxerr){
            maxerr=fabsf(x[i]-y[i])/y[i];
        }
    }
    printf("max err = %g\n",maxerr);
}
void host_mm(const double *a,const double *b,double *c){
    int i,j,k;
    for(i=0;i<10*BLK;i++){
        for(j=0;j<20*BLK;j++){
            for(k=0;k<10*BLK;k++){
                C[i][j]+=A[i][k]*B[k][j];
            }
        }
    }
}

void print(double *c){
    int i,j;
    for(i=0;i<10*BLK;i++){
        for(j=0;j<20*BLK;j++){
            printf("%.2f\t",C[i][j]);
        }
        printf("\n");
    }
}

__global__
void device_mm(const double *a,const double *b,double *c){
    int k;
    for(k=0;k<10*BLK;k++)
        C[bx*BLK+tx][by*BLK+ty]+=A[bx*BLK+tx][k]*B[k][by*BLK+ty];
}

__global__
void tiled_device_mm(const double *a,const double *b,double *c){
    __shared__ double sA[BLK][BLK];
    __shared__ double sB[BLK][BLK];
    int s,i;
    double sum=0;
    for(s=0;s<10;s++){
        sA[tx][ty]=A[bx*BLK+tx][s*BLK+ty];
        sB[tx][ty]=B[s*BLK+tx][by*BLK+ty];
        __syncthreads();
        for(i=0;i<BLK;i++){
            sum+=sA[tx][i]*sB[i][ty];
        }
        __syncthreads();
    }
    C[bx*BLK+tx][by*BLK+ty]=sum;
}



int main(){
    clock_t start,finish;
    double hosttime,devicetime;

    dim3 grid(BLK,BLK);
    dim3 block(10,20);

    double *hA,*hB,*rC,*dA,*dB,*dC,*hC;
    hA=(double*)malloc(szA);
    hB=(double*)malloc(szB);
    hC=(double*)malloc(szB);
    rC=(double*)malloc(szB);

    init(dimA,hA);
    init(dimB,hB);
    memset(hC,0,szB);

    start=clock();
    host_mm(hA,hB,hC);
    finish=clock();
    hosttime=(double)(finish-start)/CLOCKS_PER_SEC;
    printf("cpu time: %.3f\n",hosttime);
    hipMalloc(&dA,szA);
    hipMalloc(&dB,szB);
    hipMalloc(&dC,szB);
    hipMemset(dC,0,szB);    
    start=clock();
    hipMemcpy(dA,hA,szA,hipMemcpyHostToDevice);
    hipMemcpy(dB,hB,szB,hipMemcpyHostToDevice);
    device_mm<<<block,grid>>>(dA,dB,dC);
    hipDeviceSynchronize();
    hipMemcpy(rC,dC,szB,hipMemcpyDeviceToHost);
    finish=clock();    
    devicetime=(double)(finish-start)/CLOCKS_PER_SEC;
    printf("gpu time1: %.3f, speedup=%.3f\n",devicetime,hosttime/devicetime);
    hipMemset(dC,0,szB);    
    start=clock();
    hipMemcpy(dA,hA,szA,hipMemcpyHostToDevice);
    hipMemcpy(dB,hB,szB,hipMemcpyHostToDevice);
    tiled_device_mm<<<block,grid>>>(dA,dB,dC);
    hipDeviceSynchronize();
    hipMemcpy(rC,dC,szB,hipMemcpyDeviceToHost);
    finish=clock();    
    devicetime=(double)(finish-start)/CLOCKS_PER_SEC;
    printf("gpu time2: %.3f, speedup=%.3f\n",devicetime,hosttime/devicetime);

    free(hA);
    free(hB);
    free(hC);
    free(rC);
    hipFree(dA);
    hipFree(dB);
    hipFree(dC);

    return 0;
}